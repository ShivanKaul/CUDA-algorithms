#include "hip/hip_runtime.h"
#include <stdio.h>
#include "lodepng.h"
#include "gputimer.h"

#define MAX(a,b) ((a) > (b) ? a : b)

#define BLOCK_WIDTH 1000


__device__ int poolOp(unsigned char *i, int p, unsigned width)
{
  return MAX(i[p], MAX(i[p + 4], MAX(i[p + (width * 4)], i[p + (width * 4) + 4])));
}

__global__ void process(unsigned char *image,unsigned char *new_image, int NUM_THREADS, unsigned width, unsigned height)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= width * height) return;
    int pos = (8 * width * (tid / (width * 2))) + (/*2*/2 * (tid%(width * 2))); 
    pos -= tid%4;
    int ang = poolOp(image, pos, width);
    new_image[tid] = ang;
}

int main(int argc, char *argv[])
{
  GpuTimer timer;
  if (argc < 3)
  {
    printf("Incorrect arguments! Input format: ./pool <name of input png> <name of output png> \n");
    return;
  }
  
  char *input_filename = argv[1];
  char *output_filename = argv[2];

  unsigned error;
  unsigned char *image, *new_image, *gimage, *gnew_image;
  unsigned width, height;
  int NUM_THREADS;

  // Read in image
  error = lodepng_decode32_file(&image, &width, &height, input_filename);
  if (error)
    printf("Error %u in lodepng: %s\n", error, lodepng_error_text(error));

  NUM_THREADS = width * height;
  //printf("NUM_THREADS: %d, with width %d and height %d\n", NUM_THREADS, width, height);
  new_image = (unsigned char *) malloc(width * height * sizeof(unsigned char));
  hipMalloc(&gimage, 4 * width * height * sizeof(unsigned char));
  hipMalloc(&gnew_image, width * height * sizeof(unsigned char));
  hipMemcpy(gimage, image, 4 * width * height * sizeof(unsigned char), hipMemcpyHostToDevice);
  // launch the kernel
  timer.Start();
  process<<<(NUM_THREADS/BLOCK_WIDTH) + 1, BLOCK_WIDTH>>>(gimage, gnew_image, NUM_THREADS, width, height);
  timer.Stop();
  // copy back the result array to the CPU
  hipMemcpy(new_image, gnew_image, width * height * sizeof(unsigned char), hipMemcpyDeviceToHost);

  hipFree(gnew_image);
  hipFree(gimage);

  lodepng_encode32_file(output_filename, new_image, width/2, height/2);

  free(image);
  free(new_image);
  printf("Time elapsed = %g ms\n", timer.Elapsed());

  return 0;
}
